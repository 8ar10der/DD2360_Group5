
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <sys/time.h>

#define ARRAY_SIZE 10000000;


__global__ void gpu_saxpy(float a, float *x, float *y){
    int N = ARRAY_SIZE;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) 
        y[i] = a * x[i] + y[i];
}

void cpu_saxpy(float a, float *x, float *y){
    int N = ARRAY_SIZE;
    for (int i = 0; i < N; i++){
        y[i] = a * x[i] + y[i];
    }
} 

double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char const *argv[])
{
    int N = ARRAY_SIZE;
    float *x,*c_y,*g_y;
    float *d_x,*d_y;
    //localhost pointer
    x = (float*)malloc(N*sizeof(float));
    c_y = (float*)malloc(N*sizeof(float));
    g_y = (float*)malloc(N*sizeof(float));
    //CUDA device pointer
    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    //init array
    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        c_y[i] = 1.0f;
        g_y[i] = 1.0f;
    }
    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,g_y,N*sizeof(float),hipMemcpyHostToDevice);
    //saxpy
    printf("Computing SAXPY on the CPU…  ");
    double CPUTime = cpuSecond();
    cpu_saxpy(2.0f,x,c_y);
    CPUTime = cpuSecond() - CPUTime;
    printf("Done! The time of the cpu computing is: %fs\n", CPUTime);
    printf("Computing SAXPY on the GPU…  ");
    double GPUTime = cpuSecond();
    gpu_saxpy<<<(N+255)/256, 256>>>(2.0f,d_x,d_y);
    GPUTime = cpuSecond() - GPUTime;
    printf("Done! The time of the gpu computing is: %fs\n", GPUTime);
    //copy result
    hipMemcpy(g_y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);
    //comparing
    int errorCount = 0;
    printf("Comparing the output for each implementation…  ");
    for (int i = 0; i < N; i++){
        // printf("<%f,%f>",c_y[i],g_y[i]);
        if (abs(c_y[i]-g_y[i]) >= 0.5f)
            errorCount++;
    }
    if (errorCount == 0){
        printf("Correct!\n");
    } else {
        printf("Not Pass, there are %d differences.\n", errorCount);
    }
    
    //free
    hipFree(d_y);
    hipFree(d_x);
    free(g_y);
    free(c_y);
    free(x);
}
