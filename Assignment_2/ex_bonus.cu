
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <stdio.h>

/* define threads and blocks */
#define N 32//blocks = N/TPB
#define TPB 512 //threads
#define TRIALS_PER_THREAD 100 //trials per thread

//randomized cointoss
__global__ void cointoss(hiprandState *states, unsigned int* coinTot, unsigned int* inCirc) {
  //setup thread seed
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int seed = id; //different seed per thread
  double piValue = -1;
  hiprand_init(seed, id, 0, &states[id]); //start CURAND
  for(int i = 0; i < TRIALS_PER_THREAD; i++) {
    // printf("Line %d, id: %d\n", __LINE__, id);
    double x = hiprand_uniform(&states[id]);
    double y = hiprand_uniform(&states[id]);
    // printf("Line %d, id: %d, x: %f, y: %f\n", __LINE__, id, x , y);

    double throwDist = sqrt(x*x + y*y); //check throw distance
    // printf("Line %d, id: %d, throwDist: %f\n", __LINE__, id, throwDist);
    atomicAdd(coinTot, 1); //increase cointot by 1
    //checks if coin is within circle
    if(throwDist <= 1.0){
      atomicAdd(inCirc, 1); //increase inCirc by 1
    }
    piValue = 4 * (double) *inCirc / (double) *coinTot;
  }
  printf(" Pi Value %0.10f, *inCirc: %d, *coinTot:%d \n", piValue, *inCirc, *coinTot);

}



int main(){
  unsigned* coinTot; //number of times coin is in circle
  unsigned* inCirc; //number of times coin is in circle
  
  hipMalloc(&coinTot, sizeof(unsigned));
  hipMalloc(&inCirc, sizeof(unsigned));

  hipMemset(coinTot, 0, sizeof(unsigned));
  hipMemset(inCirc, 0, sizeof(unsigned));


  hiprandState *dev_random;
  hipMalloc((void**)&dev_random, TPB*sizeof(hiprandState));

  cointoss<<<TPB/N, TPB>>>(dev_random, coinTot, inCirc);
  // generates total of in and out of coinTot
  hipDeviceSynchronize();
  hipFree(dev_random);

  return 0;
}
