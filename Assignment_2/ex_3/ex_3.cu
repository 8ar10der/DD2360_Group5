
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <assert.h>
#include <sys/time.h>

#define NUM_ITERATIONS 200

typedef struct
{
    float3 position;
    float3 velocity;
} Particle;

__global__ void updateParticlesKernel(Particle *particles, unsigned totalParticles)
{
    // int threadID = threadIdx.x;
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (unsigned k = 0; k < NUM_ITERATIONS; k++)
    {
        if (i < totalParticles)
        {
            particles[i].velocity.x += 0.1;
            particles[i].velocity.y += 0.001;
            particles[i].velocity.z -= 0.002;
            particles[i].position.x += particles[i].velocity.x * 1;
        }
    }
}

__host__ void checkConsistency(Particle *particlesHost, Particle *particlesDevice, unsigned numberOfParticles)
{
    for (unsigned i = 0; i < numberOfParticles; i++)
    {
#if defined DEBUG
        printf("host: %f, device: %f\n", particlesHost[i].position.x, particlesDevice[i].position.x);
        printf("host: %f, device: %f\n", particlesHost[i].position.y, particlesDevice[i].position.y);
#endif

        assert(particlesHost[i].position.x == particlesDevice[i].position.x);
        assert(particlesHost[i].position.y == particlesDevice[i].position.y);
        assert(particlesHost[i].position.z == particlesDevice[i].position.z);

        assert(particlesHost[i].velocity.x == particlesDevice[i].velocity.x);
        assert(particlesHost[i].velocity.y == particlesDevice[i].velocity.y);
        assert(particlesHost[i].velocity.z == particlesDevice[i].velocity.z);
    }
}

__host__ void updateParticles(Particle *particles, unsigned numberOfParticles)
{

    for (unsigned k = 0; k < NUM_ITERATIONS; k++)
    {
        for (unsigned i = 0; i < numberOfParticles; i++)
        {
            particles[i].velocity.x += 0.1;
            particles[i].velocity.y += 0.001;
            particles[i].velocity.z -= 0.002;
            particles[i].position.x += particles[i].velocity.x * 1;
        }
    }
}

double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__host__ void generateRandomParticles(Particle *particles, unsigned numberOfParticles)
{
    for (unsigned i = 0; i < numberOfParticles; i++)
    {
        particles[i].position = make_float3((float)std::rand() / RAND_MAX, (float)std::rand() / RAND_MAX, (float)std::rand() / RAND_MAX);
        particles[i].velocity = make_float3((float)std::rand() / RAND_MAX, (float)std::rand() / RAND_MAX, (float)std::rand() / RAND_MAX);
    }
}

int main(int argc, char const *argv[])
{
    Particle *particles;
    Particle *cudaParticles;

    // Number of Particles
    unsigned NUM_PARTICLES = atoi(argv[1]);

    // Number of threads per block
    unsigned TBP = atoi(argv[2]);

    printf("NUM_PARTICLES: %d TBP: %d \n", NUM_PARTICLES, TBP);

    size_t particlesSize = NUM_PARTICLES * sizeof(Particle);

    particles = (Particle *)malloc(particlesSize);
    hipMalloc(&cudaParticles, particlesSize);

    Particle *cudaParticlesOnHost;
    cudaParticlesOnHost = (Particle *)malloc(particlesSize);

    generateRandomParticles(particles, NUM_PARTICLES);

    double updateParticlesKernelStart = cpuSecond();
    hipMemcpy(cudaParticles, particles, particlesSize, hipMemcpyHostToDevice);
    // <blocks, threads per block>

    unsigned BLOCKS = (NUM_PARTICLES + TBP - 1) / TBP;
    updateParticlesKernel<<<BLOCKS, TBP>>>(cudaParticles, NUM_PARTICLES);
    hipDeviceSynchronize();
    hipMemcpy(cudaParticlesOnHost, cudaParticles, particlesSize, hipMemcpyDeviceToHost);
    double updateParticlesKernelTime = cpuSecond() - updateParticlesKernelStart;

    double updateParticlesStart = cpuSecond();
    updateParticles(particles, NUM_PARTICLES);
    double updateParticlesTime = cpuSecond() - updateParticlesStart;

    printf("updateParticlesTime: %f updateParticlesKernelTime: %f \n", updateParticlesTime, updateParticlesKernelTime);

    checkConsistency(particles, cudaParticlesOnHost, NUM_PARTICLES);

    free(particles);
    free(cudaParticlesOnHost);
    hipFree(cudaParticles);
}
